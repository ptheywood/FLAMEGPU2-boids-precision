#include "hip/hip_runtime.h"
#include "flamegpu/flamegpu.h"

FLAMEGPU_AGENT_FUNCTION(outputMessage, flamegpu::MessageNone, flamegpu::MessageSpatial3D) {
    FLAMEGPU->message_out.setVariable<flamegpu::id_t>("id", FLAMEGPU->getID());
    FLAMEGPU->message_out.setLocation(
        FLAMEGPU->getVariable<float>("x"),
        FLAMEGPU->getVariable<float>("y"),
        FLAMEGPU->getVariable<float>("z"));
    return flamegpu::ALIVE;
}
FLAMEGPU_AGENT_FUNCTION(move, flamegpu::MessageSpatial3D, flamegpu::MessageNone) {
    const flamegpu::id_t ID = FLAMEGPU->getID();
    const float REPULSE_FACTOR = FLAMEGPU->environment.getProperty<float>("repulse");
    const float RADIUS = FLAMEGPU->message_in.radius();
    float fx = 0.0;
    float fy = 0.0;
    float fz = 0.0;
    const float x1 = FLAMEGPU->getVariable<float>("x");
    const float y1 = FLAMEGPU->getVariable<float>("y");
    const float z1 = FLAMEGPU->getVariable<float>("z");
    int count = 0;
    for (const auto &message : FLAMEGPU->message_in(x1, y1, z1)) {
        if (message.getVariable<flamegpu::id_t>("id") != ID) {
            const float x2 = message.getVariable<float>("x");
            const float y2 = message.getVariable<float>("y");
            const float z2 = message.getVariable<float>("z");
            float x21 = x2 - x1;
            float y21 = y2 - y1;
            float z21 = z2 - z1;
            const float separation = sqrtf(x21*x21 + y21*y21 + z21*z21);
            if (separation < RADIUS && separation > 0.0f) {
                float k = sinf((separation / RADIUS)*3.141f*-2)*REPULSE_FACTOR;
                // Normalise without recalculating separation
                x21 /= separation;
                y21 /= separation;
                z21 /= separation;
                fx += k * x21;
                fy += k * y21;
                fz += k * z21;
                count++;
            }
        }
    }
    fx /= count > 0 ? count : 1;
    fy /= count > 0 ? count : 1;
    fz /= count > 0 ? count : 1;
    FLAMEGPU->setVariable<float>("x", x1 + fx);
    FLAMEGPU->setVariable<float>("y", y1 + fy);
    FLAMEGPU->setVariable<float>("z", z1 + fz);
    FLAMEGPU->setVariable<float>("drift", sqrtf(fx*fx + fy*fy + fz*fz));
    return flamegpu::ALIVE;
}
FLAMEGPU_STEP_FUNCTION(Validation) {
    static float prevTotalDrift = FLT_MAX;
    static unsigned int driftDropped = 0;
    static unsigned int driftIncreased = 0;
    // This value should decline? as the model moves towards a steady equlibrium state
    // Once an equilibrium state is reached, it is likely to oscillate between 2-4? values
    float totalDrift = FLAMEGPU->agent("Circle").sum<float>("drift");
    if (totalDrift <= prevTotalDrift)
        driftDropped++;
    else
        driftIncreased++;
    prevTotalDrift = totalDrift;
    // printf("Avg Drift: %g\n", totalDrift / FLAMEGPU->agent("Circle").count());
    printf("%.2f%% Drift correct\n", 100 * driftDropped / static_cast<float>(driftDropped + driftIncreased));
}
int main(int argc, const char ** argv) {
    flamegpu::ModelDescription model("template");

    const unsigned int AGENT_COUNT = 16384;
    const float ENV_MAX = static_cast<float>(floor(cbrt(AGENT_COUNT)));
    const float RADIUS = 2.0f;

    // global environment variables
    flamegpu::EnvironmentDescription env = model.Environment();
    env.newProperty("repulse", 0.05f);

    // Location message
    flamegpu::MessageSpatial3D::Description message = model.newMessage<flamegpu::MessageSpatial3D>("location");
    message.newVariable<flamegpu::id_t>("id");
    message.setRadius(RADIUS);
    message.setMin(0, 0, 0);
    message.setMax(ENV_MAX, ENV_MAX, ENV_MAX);
    
    // Circle agent
    flamegpu::AgentDescription  agent = model.newAgent("Circle");
    agent.newVariable<float>("x");
    agent.newVariable<float>("y");
    agent.newVariable<float>("z");
    agent.newVariable<float>("drift");  // Store the distance moved here, for validation
    
    // Define each function. 
    flamegpu::AgentFunctionDescription outputMessageDescription = agent.newFunction("outputMessage", outputMessage);
    outputMessageDescription.setMessageOutput("location");
    flamegpu::AgentFunctionDescription moveDescription = agent.newFunction("move", move);
    moveDescription.setMessageInput("location");
    // Add a dependency that move requires outputMessage to have executed
    moveDescription.dependsOn(outputMessageDescription);

    // Identify the root of execution
    model.addExecutionRoot(outputMessageDescription);
    
    // Add a step function which in this case is used as a crude form of validation
    model.addStepFunction(Validation);

    // Build the exeuction graph
    model.generateLayers();

    // Create the simulation
    flamegpu::CUDASimulation simulation(model, argc, argv);

    // Create visualisation if enabled
#ifdef FLAMEGPU_VISUALISATION
    flamegpu::visualiser::ModelVis visualiser = simulation.getVisualisation();
    {
        const float INIT_CAM = ENV_MAX * 1.25F;
        visualiser.setInitialCameraLocation(INIT_CAM, INIT_CAM, INIT_CAM);
        visualiser.setCameraSpeed(0.01f);
        auto cirlceAgentVisualiser = visualiser.addAgent("Circle");
        // Position vars are named x, y, z; so they are used by default
        cirlceAgentVisualiser.setModel(flamegpu::visualiser::Stock::Models::ICOSPHERE);
        cirlceAgentVisualiser.setModelScale(1/10.0f);
        // Render the Subdivision of spatial messaging
        {
            const float ENV_MIN = 0;
            const int DIM = static_cast<int>(ceil((ENV_MAX - ENV_MIN) / RADIUS));  // Spatial partitioning scales up to fit none exact environments
            const float DIM_MAX = DIM * RADIUS;
            auto pen = visualiser.newLineSketch(1, 1, 1, 0.2f);  // white
            // X lines
            for (int y = 0; y <= DIM; y++) {
                for (int z = 0; z <= DIM; z++) {
                    pen.addVertex(ENV_MIN, y * RADIUS, z * RADIUS);
                    pen.addVertex(DIM_MAX, y * RADIUS, z * RADIUS);
                }
            }
            // Y axis
            for (int x = 0; x <= DIM; x++) {
                for (int z = 0; z <= DIM; z++) {
                    pen.addVertex(x * RADIUS, ENV_MIN, z * RADIUS);
                    pen.addVertex(x * RADIUS, DIM_MAX, z * RADIUS);
                }
            }
            // Z axis
            for (int x = 0; x <= DIM; x++) {
                for (int y = 0; y <= DIM; y++) {
                    pen.addVertex(x * RADIUS, y * RADIUS, ENV_MIN);
                    pen.addVertex(x * RADIUS, y * RADIUS, DIM_MAX);
                }
            }
        }
    }
    visualiser.activate();
#endif
    
    // initialise a population of agents if not provided on disk
    if (simulation.getSimulationConfig().input_file.empty()) {
        // Currently population has not been init, so generate an agent population on the fly
        std::mt19937_64 rng;
        std::uniform_real_distribution<float> dist(0.0f, ENV_MAX);
        flamegpu::AgentVector population(model.Agent("Circle"), AGENT_COUNT);
        for (unsigned int i = 0; i < AGENT_COUNT; i++) {
            flamegpu::AgentVector::Agent instance = population[i];
            instance.setVariable<float>("x", dist(rng));
            instance.setVariable<float>("y", dist(rng));
            instance.setVariable<float>("z", dist(rng));
        }
        simulation.setPopulationData(population);
    }

    // Execute the simulation
    simulation.simulate();

#ifdef FLAMEGPU_VISUALISATION
    visualiser.join();
#endif

    // Ensure profiling / memcheck work correctly
    flamegpu::util::cleanup();

    return EXIT_SUCCESS;
}
